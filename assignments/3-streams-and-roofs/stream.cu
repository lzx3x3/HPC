#include "hip/hip_runtime.h"
/*-----------------------------------------------------------------------*/
/* Program: STREAM                                                       */
/* Revision: $Id: stream.c,v 5.10 2013/01/17 16:01:06 mccalpin Exp mccalpin $ */
/* Original code developed by John D. McCalpin                           */
/* Programmers: John D. McCalpin                                         */
/*              Joe R. Zagar                                             */
/*                                                                       */
/* This program measures memory transfer rates in MB/s for simple        */
/* computational kernels coded in C.                                     */
/*-----------------------------------------------------------------------*/
/* Copyright 1991-2013: John D. McCalpin                                 */
/*-----------------------------------------------------------------------*/
/* License:                                                              */
/*  1. You are free to use this program and/or to redistribute           */
/*     this program.                                                     */
/*  2. You are free to modify this program for your own use,             */
/*     including commercial use, subject to the publication              */
/*     restrictions in item 3.                                           */
/*  3. You are free to publish results obtained from running this        */
/*     program, or from works that you derive from this program,         */
/*     with the following limitations:                                   */
/*     3a. In order to be referred to as "STREAM benchmark results",     */
/*         published results must be in conformance to the STREAM        */
/*         Run Rules, (briefly reviewed below) published at              */
/*         http://www.cs.virginia.edu/stream/ref.html                    */
/*         and incorporated herein by reference.                         */
/*         As the copyright holder, John McCalpin retains the            */
/*         right to determine conformity with the Run Rules.             */
/*     3b. Results based on modified source code or on runs not in       */
/*         accordance with the STREAM Run Rules must be clearly          */
/*         labelled whenever they are published.  Examples of            */
/*         proper labelling include:                                     */
/*           "tuned STREAM benchmark results"                            */
/*           "based on a variant of the STREAM benchmark code"           */
/*         Other comparable, clear, and reasonable labelling is          */
/*         acceptable.                                                   */
/*     3c. Submission of results to the STREAM benchmark web site        */
/*         is encouraged, but not required.                              */
/*  4. Use of this program or creation of derived works based on this    */
/*     program constitutes acceptance of these licensing restrictions.   */
/*  5. Absolutely no warranty is expressed or implied.                   */
/*-----------------------------------------------------------------------*/

/* I followed
 * https://github.com/nattoheaven/cuda_stream_benchmark/blob/master/stream.cu
 * while making my own modifications. -- Toby Isaac */
# include <stdio.h>
# include <unistd.h>
# include <math.h>
# include <float.h>
# include <limits.h>
# include <sys/time.h>

/*-----------------------------------------------------------------------
 * INSTRUCTIONS:
 *
 *	1) STREAM requires different amounts of memory to run on different
 *           systems, depending on both the system cache size(s) and the
 *           granularity of the system timer.
 *     You should adjust the value of 'STREAM_ARRAY_SIZE' (below)
 *           to meet *both* of the following criteria:
 *       (a) Each array must be at least 4 times the size of the
 *           available cache memory. I don't worry about the difference
 *           between 10^6 and 2^20, so in practice the minimum array size
 *           is about 3.8 times the cache size.
 *           Example 1: One Xeon E3 with 8 MB L3 cache
 *               STREAM_ARRAY_SIZE should be >= 4 million, giving
 *               an array size of 30.5 MB and a total memory requirement
 *               of 91.5 MB.  
 *           Example 2: Two Xeon E5's with 20 MB L3 cache each (using OpenMP)
 *               STREAM_ARRAY_SIZE should be >= 20 million, giving
 *               an array size of 153 MB and a total memory requirement
 *               of 458 MB.  
 *       (b) The size should be large enough so that the 'timing calibration'
 *           output by the program is at least 20 clock-ticks.  
 *           Example: most versions of Windows have a 10 millisecond timer
 *               granularity.  20 "ticks" at 10 ms/tic is 200 milliseconds.
 *               If the chip is capable of 10 GB/s, it moves 2 GB in 200 msec.
 *               This means the each array must be at least 1 GB, or 128M elements.
 *
 *      Version 5.10 increases the default array size from 2 million
 *          elements to 10 million elements in response to the increasing
 *          size of L3 caches.  The new default size is large enough for caches
 *          up to 20 MB. 
 *      Version 5.10 changes the loop index variables from "register int"
 *          to "ssize_t", which allows array indices >2^32 (4 billion)
 *          on properly configured 64-bit systems.  Additional compiler options
 *          (such as "-mcmodel=medium") may be required for large memory runs.
 *
 *      Array size can be set at compile time without modifying the source
 *          code for the (many) compilers that support preprocessor definitions
 *          on the compile line.  E.g.,
 *                gcc -O -DSTREAM_ARRAY_SIZE=100000000 stream.c -o stream.100M
 *          will override the default size of 10M with a new size of 100M elements
 *          per array.
 */
#ifndef STREAM_ARRAY_SIZE
#   define STREAM_ARRAY_SIZE	31000000
#endif

/*  2) STREAM runs each kernel "NTIMES" times and reports the *best* result
 *         for any iteration after the first, therefore the minimum value
 *         for NTIMES is 2.
 *      There are no rules on maximum allowable values for NTIMES, but
 *         values larger than the default are unlikely to noticeably
 *         increase the reported performance.
 *      NTIMES can also be set on the compile line without changing the source
 *         code using, for example, "-DNTIMES=7".
 */
#ifdef NTIMES
#if NTIMES<=1
#   define NTIMES	10
#endif
#endif
#ifndef NTIMES
#   define NTIMES	10
#endif

/*  Users are allowed to modify the "OFFSET" variable, which *may* change the
 *         relative alignment of the arrays (though compilers may change the 
 *         effective offset by making the arrays non-contiguous on some systems). 
 *      Use of non-zero values for OFFSET can be especially helpful if the
 *         STREAM_ARRAY_SIZE is set to a value close to a large power of 2.
 *      OFFSET can also be set on the compile line without changing the source
 *         code using, for example, "-DOFFSET=56".
 */
#ifndef OFFSET
#   define OFFSET	0
#endif

/*
 *	3) Compile the code with optimization.  Many compilers generate
 *       unreasonably bad code before the optimizer tightens things up.  
 *     If the results are unreasonably good, on the other hand, the
 *       optimizer might be too smart for me!
 *
 *     For a simple single-core version, try compiling with:
 *            cc -O stream.c -o stream
 *     This is known to work on many, many systems....
 *
 *     To use multiple cores, you need to tell the compiler to obey the OpenMP
 *       directives in the code.  This varies by compiler, but a common example is
 *            gcc -O -fopenmp stream.c -o stream_omp
 *       The environment variable OMP_NUM_THREADS allows runtime control of the 
 *         number of threads/cores used when the resulting "stream_omp" program
 *         is executed.
 *
 *     To run with single-precision variables and arithmetic, simply add
 *         -DSTREAM_TYPE=float
 *     to the compile line.
 *     Note that this changes the minimum array sizes required --- see (1) above.
 *
 *     The preprocessor directive "TUNED" does not do much -- it simply causes the 
 *       code to call separate functions to execute each kernel.  Trivial versions
 *       of these functions are provided, but they are *not* tuned -- they just 
 *       provide predefined interfaces to be replaced with tuned code.
 *
 *
 *	4) Optional: Mail the results to mccalpin@cs.virginia.edu
 *	   Be sure to include info that will help me understand:
 *		a) the computer hardware configuration (e.g., processor model, memory type)
 *		b) the compiler name/version and compilation flags
 *      c) any run-time information (such as OMP_NUM_THREADS)
 *		d) all of the output from the test case.
 *
 * Thanks!
 *
 *-----------------------------------------------------------------------*/

# define HLINE "-------------------------------------------------------------\n"

# ifndef MIN
# define MIN(x,y) ((x)<(y)?(x):(y))
# endif
# ifndef MAX
# define MAX(x,y) ((x)>(y)?(x):(y))
# endif

#ifndef STREAM_TYPE
#define STREAM_TYPE double
#endif

#define CUDA_CHK(e) do {if ((e) != hipSuccess) {printf("line %d: CUDA error: %s\n", __LINE__, hipGetErrorString(e)); return 1;}} while(0)

/* We are going to work on device memory */
static __device__ STREAM_TYPE   a[STREAM_ARRAY_SIZE+OFFSET],
                                b[STREAM_ARRAY_SIZE+OFFSET],
                                c[STREAM_ARRAY_SIZE+OFFSET];

/* notice this is a d_ symbol, indicating that there will probably be a host and a device version */
static __device__ STREAM_TYPE d_sum[3];

static double	avgtime[4] = {0}, maxtime[4] = {0},
		mintime[4] = {FLT_MAX,FLT_MAX,FLT_MAX,FLT_MAX};

static const char	*label[4] = {"Copy:      ", "Scale:     ",
    "Add:       ", "Triad:     "};

static double	bytes[4] = {
    2 * sizeof(STREAM_TYPE) * STREAM_ARRAY_SIZE,
    2 * sizeof(STREAM_TYPE) * STREAM_ARRAY_SIZE,
    3 * sizeof(STREAM_TYPE) * STREAM_ARRAY_SIZE,
    3 * sizeof(STREAM_TYPE) * STREAM_ARRAY_SIZE
    };

extern double mysecond();
extern void checkSTREAMresults(dim3 grid, dim3 block);

/* This is a launchable kernel: it has the __global__ attribute */
static __global__ void STREAM_Init_1D()
{
  /* 1D decomposition */
  int tid   = blockIdx.x * blockDim.x + threadIdx.x;
  int gSize = gridDim.x * blockDim.x;

  /* grid stride, should work for any number of thread blocks */
  for (int j = tid; j < STREAM_ARRAY_SIZE; j += gSize) {
    a[j] = 1.0;
    b[j] = 2.0;
    c[j] = 0.0;
  }
}

static __global__ void STREAM_Test_1D()
{
  /* 1D decomposition */
  int tid   = blockIdx.x * blockDim.x + threadIdx.x;
  int gSize = gridDim.x * blockDim.x;

  /* grid stride, should work for any number of thread blocks */
  for (int j = tid; j < STREAM_ARRAY_SIZE; j += gSize) {
    a[j] = 2.0 * a[j];
  }
}

static __global__ void STREAM_Copy_1D()
{
  /* 1D decomposition */
  int tid   = blockIdx.x * blockDim.x + threadIdx.x;
  int gSize = gridDim.x * blockDim.x;

  /* grid stride, should work for any number of thread blocks */
  for (int j = tid; j < STREAM_ARRAY_SIZE; j += gSize) {
    c[j] = a[j];
  }
}

static __global__ void STREAM_Scale_1D(double scalar)
{
  /* 1D decomposition */
  int tid   = blockIdx.x * blockDim.x + threadIdx.x;
  int gSize = gridDim.x * blockDim.x;

  /* grid stride, should work for any number of thread blocks */
  for (int j = tid; j < STREAM_ARRAY_SIZE; j += gSize) {
    b[j] = scalar * c[j];
  }
}

static __global__ void STREAM_Add_1D()
{
  /* 1D decomposition */
  int tid   = blockIdx.x * blockDim.x + threadIdx.x;
  int gSize = gridDim.x * blockDim.x;

  /* grid stride, should work for any number of thread blocks */
  for (int j = tid; j < STREAM_ARRAY_SIZE; j += gSize) {
    c[j] = a[j] + b[j];
  }
}

static __global__ void STREAM_Triad_1D(double scalar)
{
  /* 1D decomposition */
  int tid   = blockIdx.x * blockDim.x + threadIdx.x;
  int gSize = gridDim.x * blockDim.x;

  /* grid stride, should work for any number of thread blocks */
  for (int j = tid; j < STREAM_ARRAY_SIZE; j += gSize) {
    a[j] = b[j] + scalar * c[j];
  }
}

/* Notice that this is a __device__ function */
static __device__ void STREAM_Sum_sub(double *sum, const double *d,
                                      double *shared)
{
  /* 1D decomposition */
  int tid   = blockIdx.x * blockDim.x + threadIdx.x;
  int gSize = gridDim.x * blockDim.x;
  double  x = 0.;

  /* grid stride, should work for any number of thread blocks */
  for (int j = tid; j < STREAM_ARRAY_SIZE; j += gSize) {
    x += d[j];
  }
  /* Now I have the sum for just my values: write it in the shared array */
  shared[threadIdx.x] = x;
  /* Do a reduction within the thread block */
  for (int w = blockDim.x; w != 1;) {
    int lastw = w;

    /* divide w in half or, if w is odd, ceiling of half */
    w = (w + 1) / 2;
    __syncthreads();
    if (threadIdx.x + w < lastw) {
      x += shared[threadIdx.x + w];
      shared[threadIdx.x] = x;
    }
  }

  /* one thread per block */
  /* This strange behavior is for devices with compute capabilities < 6.x,
     otherwise we can use atomicAdd() with doubles */
  if (threadIdx.x == 0) {
    unsigned long long int *address = (unsigned long long int *)sum;
    unsigned long long int old = *address, assumed;
    do {
      assumed = old;
      old = atomicCAS(address, assumed,
	  __double_as_longlong(x + __longlong_as_double(assumed)));
    } while (assumed != old);
  }
}

static __global__ void STREAM_Sum()
{
  /* get access to the shared memory allocated at the start of the call */
  extern __shared__ double shared[];
  STREAM_Sum_sub(&d_sum[0], a, shared);
  STREAM_Sum_sub(&d_sum[1], b, shared);
  STREAM_Sum_sub(&d_sum[2], c, shared);
}

int
main()
    {
    int     nDevice;
    int			quantum, checktick();
    int			BytesPerWord;
    int			k;
    ssize_t		j;
    STREAM_TYPE		scalar;
    double		t, times[4][NTIMES];
    dim3                grid, block;
    hipError_t         err;

    /* --- SETUP --- determine precision and check timing --- */

    printf(HLINE);
    printf("CSE6230 CUDA STREAM based on version $Revision: 5.10 $\n");
    printf(HLINE);
    BytesPerWord = sizeof(STREAM_TYPE);
    printf("This system uses %d bytes per array element.\n",
	BytesPerWord);

    printf(HLINE);
#ifdef N
    printf("*****  WARNING: ******\n");
    printf("      It appears that you set the preprocessor variable N when compiling this code.\n");
    printf("      This version of the code uses the preprocesor variable STREAM_ARRAY_SIZE to control the array size\n");
    printf("      Reverting to default value of STREAM_ARRAY_SIZE=%llu\n",(unsigned long long) STREAM_ARRAY_SIZE);
    printf("*****  WARNING: ******\n");
#endif

    printf("Array size = %llu (elements), Offset = %d (elements)\n" , (unsigned long long) STREAM_ARRAY_SIZE, OFFSET);
    printf("Memory per array = %.1f MiB (= %.1f GiB).\n", 
	BytesPerWord * ( (double) STREAM_ARRAY_SIZE / 1024.0/1024.0),
	BytesPerWord * ( (double) STREAM_ARRAY_SIZE / 1024.0/1024.0/1024.0));
    printf("Total memory required = %.1f MiB (= %.1f GiB).\n",
	(3.0 * BytesPerWord) * ( (double) STREAM_ARRAY_SIZE / 1024.0/1024.),
	(3.0 * BytesPerWord) * ( (double) STREAM_ARRAY_SIZE / 1024.0/1024./1024.));
    printf("Each kernel will be executed %d times.\n", NTIMES);
    printf(" The *best* time for each kernel (excluding the first iteration)\n"); 
    printf(" will be used to compute the reported bandwidth.\n");

    /* Get initial value for system clock. */
    {
      int k;
      int x;
      struct hipDeviceProp_t prop;
      err = hipGetDeviceCount(&nDevice); CUDA_CHK(err);
      for (k = 0; k < nDevice; k++) {
        err = hipGetDeviceProperties(&prop, k); CUDA_CHK(err);
        printf("Device Number: %d\n", k);
        printf("  Device name: %s\n", prop.name);
        printf("  Memory Clock Rate (KHz): %d\n", prop.memoryClockRate);
        printf("  Memory Bus Width (bits): %d\n", prop.memoryBusWidth);
        printf("  Peak Memory Bandwidth (GB/s): %f\n\n", 2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);
      }
      err = hipGetDevice(&k); CUDA_CHK(err);
      printf ("Ordinal of GPUs requested = %i\n",k);
      err = hipGetDeviceProperties(&prop, k); CUDA_CHK(err);
      block = dim3(prop.maxThreadsPerBlock);
      x = (STREAM_ARRAY_SIZE + block.x - 1) / block.x;
      grid = dim3(x);
    }

    printf(HLINE);

    /* Get initial value for system clock. */
    STREAM_Init_1D<<<grid, block>>>();

    printf(HLINE);

    if  ( (quantum = checktick()) >= 1) 
	printf("Your clock granularity/precision appears to be "
	    "%d microseconds.\n", quantum);
    else {
	printf("Your clock granularity appears to be "
	    "less than one microsecond.\n");
	quantum = 1;
    }

    t = mysecond();
    STREAM_Test_1D<<<grid, block>>>();
    err = hipDeviceSynchronize(); CUDA_CHK(err);
    t = 1.0E6 * (mysecond() - t);

    printf("Each test below will take on the order"
	" of %d microseconds.\n", (int) t  );
    printf("   (= %d clock ticks)\n", (int) (t/quantum) );
    printf("Increase the size of the arrays if this shows that\n");
    printf("you are not getting at least 20 clock ticks per test.\n");

    printf(HLINE);

    printf("WARNING -- The above is only a rough guideline.\n");
    printf("For best results, please be sure you know the\n");
    printf("precision of your system timer.\n");
    printf(HLINE);
    
    /*	--- MAIN LOOP --- repeat test cases NTIMES times --- */

    scalar = 3.0;
    for (k=0; k<NTIMES; k++)
	{
	times[0][k] = mysecond();
        STREAM_Copy_1D<<<grid, block>>>();
	err = hipDeviceSynchronize(); CUDA_CHK(err);
	times[0][k] = mysecond() - times[0][k];
	
	times[1][k] = mysecond();
        STREAM_Scale_1D<<<grid, block>>>(scalar);
	err = hipDeviceSynchronize(); CUDA_CHK(err);
	times[1][k] = mysecond() - times[1][k];
	
	times[2][k] = mysecond();
        STREAM_Add_1D<<<grid, block>>>();
	err = hipDeviceSynchronize(); CUDA_CHK(err);
	times[2][k] = mysecond() - times[2][k];
	
	times[3][k] = mysecond();
        STREAM_Triad_1D<<<grid, block>>>(scalar);
	err = hipDeviceSynchronize(); CUDA_CHK(err);
	times[3][k] = mysecond() - times[3][k];
	}

    /*	--- SUMMARY --- */

    for (k=1; k<NTIMES; k++) /* note -- skip first iteration */
	{
	for (j=0; j<4; j++)
	    {
	    avgtime[j] = avgtime[j] + times[j][k];
	    mintime[j] = MIN(mintime[j], times[j][k]);
	    maxtime[j] = MAX(maxtime[j], times[j][k]);
	    }
	}
    
    printf("Function    Best Rate MB/s  Avg time     Min time     Max time\n");
    for (j=0; j<4; j++) {
		avgtime[j] = avgtime[j]/(double)(NTIMES-1);

		printf("%s%12.1f  %11.6f  %11.6f  %11.6f\n", label[j],
	       1.0E-06 * bytes[j]/mintime[j],
	       avgtime[j],
	       mintime[j],
	       maxtime[j]);
    }
    printf(HLINE);

    /* --- Check Results --- */
    checkSTREAMresults(grid, block);
    printf(HLINE);

    return 0;
}

# define	M	20

int
checktick()
    {
    int		i, minDelta, Delta;
    double	t1, t2, timesfound[M];

/*  Collect a sequence of M unique time values from the system. */

    for (i = 0; i < M; i++) {
	t1 = mysecond();
	while( ((t2=mysecond()) - t1) < 1.0E-6 )
	    ;
	timesfound[i] = t1 = t2;
	}

/*
 * Determine the minimum difference between these M values.
 * This result will be our estimate (in microseconds) for the
 * clock granularity.
 */

    minDelta = 1000000;
    for (i = 1; i < M; i++) {
	Delta = (int)( 1.0E6 * (timesfound[i]-timesfound[i-1]));
	minDelta = MIN(minDelta, MAX(Delta,0));
	}

   return(minDelta);
    }



/* A gettimeofday routine to give access to the wall
   clock timer on most UNIX-like systems.  */

#include <sys/time.h>

double mysecond()
{
        struct timeval tp;
        struct timezone tzp;

        (void) gettimeofday(&tp,&tzp);
        return ( (double) tp.tv_sec + (double) tp.tv_usec * 1.e-6 );
}

#ifndef abs
#define abs(a) ((a) >= 0 ? (a) : -(a))
#endif
void checkSTREAMresults (dim3 grid, dim3 block)
{
	STREAM_TYPE aj,bj,cj,scalar;
	double h_sum[3];
	double epsilon;
	int	k,serr;

    /* reproduce initialization */
	aj = 1.0;
	bj = 2.0;
	cj = 0.0;
    /* a[] is modified during timing check */
	aj = 2.0E0 * aj;
    /* now execute timing loop */
	scalar = 3.0;
	for (k=0; k<NTIMES; k++)
        {
            cj = aj;
            bj = scalar*cj;
            cj = aj+bj;
            aj = bj+scalar*cj;
        }

	aj = aj * (double) (STREAM_ARRAY_SIZE);
	bj = bj * (double) (STREAM_ARRAY_SIZE);
	cj = cj * (double) (STREAM_ARRAY_SIZE);

	h_sum[0] = 0.0;
	h_sum[1] = 0.0;
	h_sum[2] = 0.0;
	hipMemcpyToSymbol(HIP_SYMBOL(d_sum), h_sum, 3 * sizeof(double));
	STREAM_Sum<<<grid, block, block.x * sizeof(double)>>>();
	hipMemcpyFromSymbol(h_sum, HIP_SYMBOL(d_sum), 3 * sizeof(double));

	if (sizeof(STREAM_TYPE) == 4) {
		epsilon = 1.e-6;
	}
	else if (sizeof(STREAM_TYPE) == 8) {
		epsilon = 1.e-12;
	}
	else {
		printf("WEIRD: sizeof(STREAM_TYPE) = %lu\n",sizeof(STREAM_TYPE));
		epsilon = 1.e-6;
	}
	serr = 0;
	if (abs(aj-h_sum[0])/h_sum[0] > epsilon) {
	  serr++;
	  printf ("Failed Validation on array a[]\n");
	  printf ("        Expected  : %f \n",aj);
	  printf ("        Observed  : %f \n",h_sum[0]);
	}
	if (abs(bj-h_sum[1])/h_sum[1] > epsilon) {
	  serr++;
	  printf ("Failed Validation on array b[]\n");
	  printf ("        Expected  : %f \n",bj);
	  printf ("        Observed  : %f \n",h_sum[1]);
	}
	if (abs(cj-h_sum[2])/h_sum[2] > epsilon) {
	  serr++;
	  printf ("Failed Validation on array c[]\n");
	  printf ("        Expected  : %f \n",cj);
	  printf ("        Observed  : %f \n",h_sum[2]);
	}

	if (serr == 0) {
		printf ("Solution Validates: avg error less than %e on all three arrays\n",epsilon);
	}
#ifdef VERBOSE
	printf ("Results Validation Verbose Results: \n");
	printf ("    Expected a(1), b(1), c(1): %f %f %f \n",aj,bj,cj);
	printf ("    Observed a(1), b(1), c(1): %f %f %f \n",a[1],b[1],c[1]);
	printf ("    Rel Errors on a, b, c:     %e %e %e \n",abs(aAvgErr/aj),abs(bAvgErr/bj),abs(cAvgErr/cj));
#endif
}
